#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/Generator.h>
#include <ATen/Tensor.h>
#include <ATen/native/DistributionTemplates.h>
#include <ATen/core/op_registration/op_registration.h>
#include <ATen/cuda/Exceptions.h>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/core/MT19937RNGEngine.h>
#include <memory>

using namespace at;

struct CUDA_CSPRNG_GeneratorImpl : public at::CPUGeneratorImpl {
  CUDA_CSPRNG_GeneratorImpl(uint64_t seed_in = default_rng_seed_val) : CPUGeneratorImpl(seed_in) {
    this->key_set_ = DispatchKeySet(DispatchKey::CustomRNGKeyId);
  }
};

typedef ulonglong2 block_t;
constexpr size_t block_t_size = sizeof(block_t);

// =========================================================== AES ===========================================================

__device__ static uint8_t sbox[256] = {
    0x63, 0x7C, 0x77, 0x7B, 0xF2, 0x6B, 0x6F, 0xC5, 0x30, 0x01, 0x67, 0x2B, 0xFE, 0xD7, 0xAB, 0x76,
    0xCA, 0x82, 0xC9, 0x7D, 0xFA, 0x59, 0x47, 0xF0, 0xAD, 0xD4, 0xA2, 0xAF, 0x9C, 0xA4, 0x72, 0xC0,
    0xB7, 0xFD, 0x93, 0x26, 0x36, 0x3F, 0xF7, 0xCC, 0x34, 0xA5, 0xE5, 0xF1, 0x71, 0xD8, 0x31, 0x15,
    0x04, 0xC7, 0x23, 0xC3, 0x18, 0x96, 0x05, 0x9A, 0x07, 0x12, 0x80, 0xE2, 0xEB, 0x27, 0xB2, 0x75,
    0x09, 0x83, 0x2C, 0x1A, 0x1B, 0x6E, 0x5A, 0xA0, 0x52, 0x3B, 0xD6, 0xB3, 0x29, 0xE3, 0x2F, 0x84,
    0x53, 0xD1, 0x00, 0xED, 0x20, 0xFC, 0xB1, 0x5B, 0x6A, 0xCB, 0xBE, 0x39, 0x4A, 0x4C, 0x58, 0xCF,
    0xD0, 0xEF, 0xAA, 0xFB, 0x43, 0x4D, 0x33, 0x85, 0x45, 0xF9, 0x02, 0x7F, 0x50, 0x3C, 0x9F, 0xA8,
    0x51, 0xA3, 0x40, 0x8F, 0x92, 0x9D, 0x38, 0xF5, 0xBC, 0xB6, 0xDA, 0x21, 0x10, 0xFF, 0xF3, 0xD2,
    0xCD, 0x0C, 0x13, 0xEC, 0x5F, 0x97, 0x44, 0x17, 0xC4, 0xA7, 0x7E, 0x3D, 0x64, 0x5D, 0x19, 0x73,
    0x60, 0x81, 0x4F, 0xDC, 0x22, 0x2A, 0x90, 0x88, 0x46, 0xEE, 0xB8, 0x14, 0xDE, 0x5E, 0x0B, 0xDB,
    0xE0, 0x32, 0x3A, 0x0A, 0x49, 0x06, 0x24, 0x5C, 0xC2, 0xD3, 0xAC, 0x62, 0x91, 0x95, 0xE4, 0x79,
    0xE7, 0xC8, 0x37, 0x6D, 0x8D, 0xD5, 0x4E, 0xA9, 0x6C, 0x56, 0xF4, 0xEA, 0x65, 0x7A, 0xAE, 0x08,
    0xBA, 0x78, 0x25, 0x2E, 0x1C, 0xA6, 0xB4, 0xC6, 0xE8, 0xDD, 0x74, 0x1F, 0x4B, 0xBD, 0x8B, 0x8A,
    0x70, 0x3E, 0xB5, 0x66, 0x48, 0x03, 0xF6, 0x0E, 0x61, 0x35, 0x57, 0xB9, 0x86, 0xC1, 0x1D, 0x9E,
    0xE1, 0xF8, 0x98, 0x11, 0x69, 0xD9, 0x8E, 0x94, 0x9B, 0x1E, 0x87, 0xE9, 0xCE, 0x55, 0x28, 0xDF,
    0x8C, 0xA1, 0x89, 0x0D, 0xBF, 0xE6, 0x42, 0x68, 0x41, 0x99, 0x2D, 0x0F, 0xB0, 0x54, 0xBB, 0x16
 };

//reduce the size of this since we don't need this many
__device__ static uint8_t rcon[255] = {
  0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 
  0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 
  0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 
  0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 
  0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 
  0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 
  0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 
  0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 
  0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 
  0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 
  0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 
  0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 
  0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 
  0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 
  0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 
  0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb  };

__device__ void add_round_key(uint8_t *block, uint8_t *key){
  for (int i = 0; i < 16; ++i) {
    block[i] = block[i] ^ key[i];
  }
}

__device__ void mix_columns(uint8_t *block){
  for (int i = 0; i < 4; ++i) { //iterate over columns
    uint8_t a[4];
    uint8_t b[4]; 
    uint8_t h;
    for (int j = 0; j < 4; ++j) {
      a[j] = block[4*i + j];
      h = (uint8_t)((int8_t)a[j] >> 7);
      b[j] = a[j] << 1;
      b[j] ^= 0x1b & h;
    } 
    block[4*i + 0] = b[0] ^ a[3] ^ a[2] ^ b[1] ^ a[1];
    block[4*i + 1] = b[1] ^ a[0] ^ a[3] ^ b[2] ^ a[2];
    block[4*i + 2] = b[2] ^ a[1] ^ a[0] ^ b[3] ^ a[3];
    block[4*i + 3] = b[3] ^ a[2] ^ a[1] ^ b[0] ^ a[0]; 
  }
}

__device__ void sub_bytes(uint8_t *block){
  for (int i = 0; i < 16; ++i) {
    block[i] = sbox[block[i]];
  }
}

//shift left by 0,1,2,3 respectively 
__device__ void shift_rows(uint8_t *block) {
  uint8_t tmp;
  //row 0 remains unshifted

  //shift row 1 left by 1
  tmp = block[1];
  block[1] = block[5];
  block[5] = block[9];
  block[9] = block[13];
  block[13] = tmp;

  //shift row 2 letf by 2
  tmp = block[2];
  block[2] = block[10];
  block[10] = tmp;

  tmp = block[6];
  block[6] = block[14];
  block[14] = tmp;

  //shift row 3 left by 3
  tmp = block[3];
  block[3] = block[15];
  block[15] = block[11];
  block[11] = block[7];
  block[7] = tmp;
}

//generate round keys from initial key
__device__ void expand_key(uint8_t *key, uint8_t *rkey){
  uint32_t i,j,k;
  uint8_t tempa[4];
  uint32_t nround = 10;
  //first round key is just the key
  for (i = 0; i < 4; ++i) {
    rkey[4*i + 0] = key[4*i + 0];
    rkey[4*i + 1] = key[4*i + 1];
    rkey[4*i + 2] = key[4*i + 2];
    rkey[4*i + 3] = key[4*i + 3];
  }
  for (i = 4; i < 4*(nround + 1); ++i) {
    for (j = 0; j < 4; ++j) {
      tempa[j] = rkey[(i-1)*4 + j];
    }
    if (i % 4 == 0) {
      //rotate 4 bytes in word
      k = tempa[0];
      tempa[0] = tempa[1];
      tempa[1] = tempa[2];
      tempa[2] = tempa[3];
      tempa[3] = k;

      tempa[0] = sbox[tempa[0]];
      tempa[1] = sbox[tempa[1]];
      tempa[2] = sbox[tempa[2]];
      tempa[3] = sbox[tempa[3]];
  
      tempa[0] = tempa[0] ^ rcon[i/4];
    }
    rkey[4*i + 0] = rkey[4*(i-4) + 0] ^ tempa[0];
    rkey[4*i + 1] = rkey[4*(i-4) + 1] ^ tempa[1];
    rkey[4*i + 2] = rkey[4*(i-4) + 2] ^ tempa[2];
    rkey[4*i + 3] = rkey[4*(i-4) + 3] ^ tempa[3];
  } 
}

__device__ void encrypt(uint8_t *block, uint8_t *key) {
  uint8_t rkey[176];
  uint8_t round; 
  //setup round keys
  expand_key(key, rkey); 
  //perform encryption
  add_round_key(block, rkey);
  for(round = 1; round < 10; ++round){
    sub_bytes(block);
    shift_rows(block);
    mix_columns(block);
    add_round_key(block, rkey + 16*round);
  }
  sub_bytes(block);
  shift_rows(block);
  add_round_key(block, rkey + 16*round);
}

// ===========================================================================================================================

template<typename scalar_t, typename uint_t, typename cipher_t, typename transform_t>
__global__ void block_cipher_contiguous_kernel(scalar_t* data, int numel, cipher_t cipher, transform_t transform_func) {
  const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  constexpr auto unroll_factor = block_t_size / sizeof(uint_t);
  if (unroll_factor * idx < numel) {
    auto block = cipher(idx);
    #pragma unroll
    for (auto i = 0; i < unroll_factor; ++i) {
      const auto li = unroll_factor * idx + i;
      if (li < numel) {
        data[li] = transform_func((reinterpret_cast<uint_t*>(&block))[i]);
      }
    }
  }
}

template<typename scalar_t, typename uint_t, typename cipher_t, typename transform_t>
__global__ void block_cipher_kernel(scalar_t* data, int numel, cipher_t cipher, transform_t transform_func, OffsetCalculator<1> offset_calc) {
  const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  constexpr auto unroll_factor = block_t_size / sizeof(uint_t);
  if (unroll_factor * idx < numel) {
    auto block = cipher(idx);
    #pragma unroll
    for (auto i = 0; i < unroll_factor; ++i) {
      const auto li = unroll_factor * idx + i;
      if (li < numel) {
        const auto offsets = offset_calc.get(li);
        data[offsets[0] / sizeof(scalar_t)] = transform_func((reinterpret_cast<uint_t*>(&block))[i]);
      }
    }
  }
}

template<typename scalar_t, typename uint_t, typename cipher_t, typename transform_t>
void block_cipher_ctr_mode(at::TensorIterator& iter, cipher_t cipher, transform_t transform_func) {
  const auto numel = iter.numel();
  if (numel == 0) {
    return;
  }
  constexpr auto unroll_factor = block_t_size / sizeof(uint_t);
  const auto block = 256;
  const auto grid = (numel + (block * unroll_factor) - 1) / (block * unroll_factor);
  scalar_t* data = (scalar_t*)iter.data_ptr(0);
  auto stream = at::cuda::getCurrentCUDAStream();
  if (iter.output(0).is_contiguous()) {
    block_cipher_contiguous_kernel<scalar_t, uint_t, cipher_t, transform_t><<<grid, block, 0, stream>>>(data, numel, cipher, transform_func);
  } else {
    auto offset_calc = make_offset_calculator<1>(iter);
    block_cipher_kernel<scalar_t, uint_t, cipher_t, transform_t><<<grid, block, 0, stream>>>(data, numel, cipher, transform_func, offset_calc);
  }
  AT_CUDA_CHECK(hipGetLastError());
}

// ===========================================================================================================================

template<typename scalar_t, typename uint_t, typename transform_t>
void random_kernel_helper(TensorIterator& iter, uint8_t* key, transform_t transform_func) {
  block_cipher_ctr_mode<scalar_t, uint_t>(iter,
    [key] __device__ (unsigned int idx) -> block_t {
      block_t block;
      memset(&block, 0, block_t_size);
      *(reinterpret_cast<unsigned int*>(&block)) = idx;
      encrypt(reinterpret_cast<uint8_t*>(&block), key);
      return block;
    },
    transform_func
  );
}

// ===========================================================================================================================

template<typename scalar_t, typename uint_t>
void random_kernel_helper_fp(TensorIterator& iter, uint8_t* key) {
  random_kernel_helper<scalar_t, uint_t>(iter, key,
    [] __device__ (uint_t rand) -> scalar_t {
      return static_cast<scalar_t>(rand % static_cast<uint64_t>((1ULL << std::numeric_limits<scalar_t>::digits) + 1));
    }
  );
}

template<typename scalar_t, typename uint_t>
void random_kernel_helper_int(TensorIterator& iter, uint8_t* key) {
  random_kernel_helper<scalar_t, uint_t>(iter, key,
    [] __device__ (uint_t rand) -> scalar_t {
      return static_cast<scalar_t>(rand % (static_cast<uint64_t>(std::numeric_limits<scalar_t>::max()) + 1));
    }
  );
}

void random_kernel_helper_bool(TensorIterator& iter, uint8_t* key) {
  random_kernel_helper<bool, uint32_t>(iter, key,
    [] __device__ (uint32_t rand) -> bool {
      return static_cast<bool>(rand & 1);
    }
  );
}

void random_kernel(TensorIterator& iter, Tensor key_tensor) {
  const auto key = key_tensor.data_ptr<uint8_t>();
  if (isFloatingType(iter.dtype())) {
    AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "random_kernel_fp_cuda", [&] {
      if (std::is_same<scalar_t, double>::value) {
        random_kernel_helper_fp<scalar_t, uint64_t>(iter, key);
      } else {
        random_kernel_helper_fp<scalar_t, uint32_t>(iter, key);
      }
    });
  } else if (isIntegralType(iter.dtype(), /*includeBool=*/true)) {
    AT_DISPATCH_INTEGRAL_TYPES_AND(at::ScalarType::Bool, iter.dtype(), "random_kernel_int_cuda", [&] {
      if (std::is_same<scalar_t, int64_t>::value) {
        random_kernel_helper_int<scalar_t, uint64_t>(iter, key);
      } else if (std::is_same<scalar_t, bool>::value) {
        random_kernel_helper_bool(iter, key);
      } else {
        random_kernel_helper_int<scalar_t, uint32_t>(iter, key);
      }
    });
  }
}

template<typename RNG>
struct RandomKernel {
  void operator()(TensorIterator& iter, Generator generator) {
    const auto key_tensor = torch::empty({16}, torch::kUInt8).random_(0, 256, generator).to(kCUDA);
    random_kernel(iter, key_tensor);
  }
};

Tensor& random_(Tensor& self, Generator generator) {
  return native::templates::random_impl<RandomKernel, CUDA_CSPRNG_GeneratorImpl>(self, generator);
}

// ===========================================================================================================================

template<typename scalar_t, typename uint_t>
void uniform_kernel_helper_fp(TensorIterator& iter, uint8_t* key, scalar_t from, scalar_t to) {
  constexpr uint_t SCALAR_T_MASK = (static_cast<uint64_t>(1) << std::numeric_limits<scalar_t>::digits) - 1;
  constexpr scalar_t SCALAR_T_DIVISOR = static_cast<scalar_t>(1) / (1ULL << std::numeric_limits<scalar_t>::digits);
  random_kernel_helper<scalar_t, uint_t>(iter, key,
    [from, to] __device__ (uint_t rand) -> scalar_t {
      return ((rand & SCALAR_T_MASK) * SCALAR_T_DIVISOR) * (to - from) + from;
    }
  );
}

void uniform_kernel(TensorIterator& iter, Tensor key_tensor, double from, double to) {
  const auto key = key_tensor.data_ptr<uint8_t>();
  AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "uniform_kernel_cuda", [&] {
    if (std::is_same<scalar_t, double>::value) {
      uniform_kernel_helper_fp<scalar_t, uint64_t>(iter, key, from, to);
    } else {
      uniform_kernel_helper_fp<scalar_t, uint32_t>(iter, key, from, to);
    }
  });
}

template<typename RNG>
struct UniformKernel {
  void operator()(TensorIterator& iter, double from, double to, Generator generator) {
    const auto key_tensor = torch::empty({16}, torch::kUInt8).random_(0, 256, generator).to(kCUDA);
    uniform_kernel(iter, key_tensor, from, to);
  }
};

Tensor& uniform_(Tensor& self, double from, double to, Generator generator) {
  return at::native::templates::uniform_impl_<UniformKernel, CUDA_CSPRNG_GeneratorImpl>(self, from, to, generator);
}

// ===========================================================================================================================

template<typename RNG>
void normal_kernel(Tensor& self, double mean_, double std_, RNG gen) {
  // TODO
}

template<typename RNG>
struct NormalKernel {
  void operator()(Tensor& self, double mean, double std, Generator gen) {
    normal_kernel(self, mean, std, check_generator<RNG>(gen));
  }
};

Tensor& normal_(Tensor& self, double mean, double std, Generator gen) {
  return at::native::templates::normal_impl_<NormalKernel, CUDA_CSPRNG_GeneratorImpl>(self, mean, std, gen);
}

// ===========================================================================================================================

Generator create_CUDA_CSPRNG_Generator() {
  return make_generator<CUDA_CSPRNG_GeneratorImpl>();
}

void registerOps() {
  static auto registry = torch::import()
    .impl_UNBOXED("aten::random_", DispatchKey::CustomRNGKeyId, random_)
    .impl_UNBOXED("aten::uniform_", DispatchKey::CustomRNGKeyId, uniform_)
    .impl_UNBOXED("aten::normal_", DispatchKey::CustomRNGKeyId, normal_);
}
  
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("registerOps", &registerOps);
  m.def("create_CUDA_CSPRNG_Generator", &create_CUDA_CSPRNG_Generator);
}
